#include "hip/hip_runtime.h"
//////////////////////////////////////////////////////////////////////////////////
//                                                                           	//
//	cleap                                                                   //
//	A library for handling / processing / rendering 3D meshes.	        //
//                                                                           	//
//////////////////////////////////////////////////////////////////////////////////
//										//
//	Copyright © 2011 Cristobal A. Navarro.					//
//										//
//	This file is part of cleap.						//
//	cleap is free software: you can redistribute it and/or modify		//
//	it under the terms of the GNU General Public License as published by	//
//	the Free Software Foundation, either version 3 of the License, or	//
//	(at your option) any later version.					//
//										//
//	cleap is distributed in the hope that it will be useful,		//
//	but WITHOUT ANY WARRANTY; without even the implied warranty of		//
//	MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the	    	//
//	GNU General Public License for more details.				//
//										//
//	You should have received a copy of the GNU General Public License	//
//	along with cleap.  If not, see <http://www.gnu.org/licenses/>. 		//
//										//
//////////////////////////////////////////////////////////////////////////////////

#ifndef _CLEAP_KERNEL_MOVE_VERTICES_H
#define _CLEAP_KERNEL_MOVE_VERTICES_H

#include "cleap_kernel_utils.cu"

__global__ void cleap_kernel_move_mesh(int vertex_count, float4* mesh_data, float2* vel, float2 min_p, float2 max_p){
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    if( i<vertex_count && i>4){
        if(mesh_data[i].x>max_p.x || mesh_data[i].x<min_p.x)vel[i].x*=-1;
        if(mesh_data[i].y>max_p.y || mesh_data[i].y<min_p.y)vel[i].y*=-1;
        mesh_data[i].x+=vel[i].x;
        mesh_data[i].y+=vel[i].y;
    }
}

__global__ void cleap_kernel_correct_overlaps(unsigned int edge_count, GLuint* triangles, float4* mesh_data, int2* edge_idx, int* listo, float radius, float2* vel){
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int stride = blockDim.x * gridDim.x;

    for(; i < edge_count; i += stride)
	{
		int2 edge = edge_idx[i];
		float4 bi = mesh_data[triangles[edge_idx[i].x]];
		float4 bj = mesh_data[triangles[edge_idx[i].y]];

		float2 r = distVec(bi,bj);
		double dist_sqrt = dot(r,r);

		if (dist_sqrt < 4*radius*radius)
		{
			// we have a collision
			listo[0] = 0;

			float2 p0 = make_float2(bi.x,bi.y);
			float2 v0 = vel[triangles[edge.x]];
			float2 p1 = make_float2(bj.x,bj.y);
			float2 v1 = vel[triangles[edge.y]];

			float2 contactNormal = normalize(p0-p1);
			float2 contactNormalNN = contactNormal*(2*radius-length(p0-p1));
			float separationVelocity = dot(v0-v1,contactNormal);
			float delta_velocity = -0.5*separationVelocity;
			float impulse = delta_velocity*2; // asumming mass = 1 for every particle
			float2 imp_per_mass = contactNormal*impulse;

			atomicAdd(&(vel[triangles[edge.x]].x),imp_per_mass.x);
			atomicAdd(&(vel[triangles[edge.x]].y),imp_per_mass.y);

			atomicAdd(&(vel[triangles[edge.y]].x),-imp_per_mass.x);
			atomicAdd(&(vel[triangles[edge.y]].y),-imp_per_mass.y);

			atomicAdd(&(mesh_data[triangles[edge.x]].x),contactNormalNN.x);
			atomicAdd(&(mesh_data[triangles[edge.x]].y),contactNormalNN.y);

			atomicAdd(&(mesh_data[triangles[edge.y]].x),-contactNormalNN.x);
			atomicAdd(&(mesh_data[triangles[edge.y]].y),-contactNormalNN.y);
		}
	}
}

#endif _CLEAP_KERNEL_MOVE_VERTICES_H
